#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "utils.h"

#define NMAX		(1 << 20)

/**
 * ~TODO 3~
 * Modify the kernel below such as each element of the 
 * array will be now equal to 0 if it is an even number
 * or 1, if it is an odd number
 */
__global__ void kernel_parity_id(int *a, int N) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N)
	{
		a[i] %= 2;
	}
}

/**
 * ~TODO 4~
 * Modify the kernel below such as each element will
 * be equal to the BLOCK ID this computation takes
 * place.
 */
__global__ void kernel_block_id(int *a, int N) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N)
	{
		a[i] = blockIdx.x;
	}
}

/**
 * ~TODO 5~
 * Modify the kernel below such as each element will
 * be equal to the THREAD ID this computation takes
 * place.
 */ 
__global__ void kernel_thread_id(int *a, int N) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N)
	{
		a[i] = threadIdx.x;
	}
}

int main(void) {
    int nDevices;
	hipDeviceProp_t prop;
	hipError_t err;
	int i;
	int* host_a;
	int* device_a;

    // Get the number of CUDA-capable GPU(s)
    hipGetDeviceCount(&nDevices);

    /**
     * ~TODO 1~
     * For each device, show some details in the format below, 
     * then set as active device the first one (assuming there
     * is at least CUDA-capable device). Pay attention to the
     * type of the fields in the hipDeviceProp_t structure.
     *
     * Device number: <i>
     *      Device name: <name>
     *      Total memory: <mem>
     *      Memory Clock Rate (KHz): <mcr>
     *      Memory Bus Width (bits): <mbw>
     * 
     * Hint: look for hipGetDeviceProperties and hipSetDevice in
     * the Cuda Toolkit Documentation. 
     */
	for (i = 0; i < nDevices; ++i) {
		hipGetDeviceProperties(&prop, i);

		printf("Device number: %d\n", i);
		printf("\tDevice name: %s\n", prop.name);
		printf("\tTotal memory: %zu\n", prop.totalGlobalMem);
		printf("\tMemory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		printf("\tMemory Bus Width (bits): %d\n", prop.memoryBusWidth);
	}


    /**
     * ~TODO 2~
     * With information from example_2.cu, allocate an array with
     * integers (where a[i] = i). Then, modify the three kernels
     * above and execute them using 4 blocks, each with 4 threads.
     *
     * You can use the fill_array(int *a, int n) function (from utils)
     * to fill your array as many times you want.
     * 
     *  ~TODO 3~
     * Execute kernel_parity_id kernel and then copy from
     * the device to the host; call hipDeviceSynchronize()
     * after a kernel execution for safety purposes.
     */ 

    host_a = (int*)malloc(NMAX * sizeof(int *));
    if (!host_a) {
        perror("malloc host_a");
        exit(1);
    }

	err = hipMalloc(&device_a, NMAX * sizeof(*device_a));
    if (err != hipSuccess) {
        perror("hipMalloc device_a");
        exit(1);
    }

	fill_array_int(host_a, NMAX);

	err = hipMemcpy(device_a, host_a, NMAX * sizeof(*host_a),
		hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        perror("hipMemcpy host_a -> device_a");
        exit(1);
    }

    kernel_parity_id<<<NMAX / 4, 4>>>(device_a, NMAX);

	err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        perror("hipDeviceSynchronize");
        exit(1);
    }

	err = hipMemcpy(host_a, device_a, NMAX * sizeof(*host_a),
		hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        perror("hipMemcpy device_a -> host_a");
        exit(1);
    }

    // Uncomment the line below to check your results
    check_task_2(3, host_a);

    /**
     * ~TODO 4~
     * Execute kernel_block_id kernel and then copy from 
     * the device to the host;
     */

    kernel_block_id<<<NMAX / 4, 4>>>(device_a, NMAX);

	err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        perror("hipDeviceSynchronize");
        exit(1);
    }

	err = hipMemcpy(host_a, device_a, NMAX * sizeof(*host_a),
		hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        perror("hipMemcpy device_a -> host_a");
        exit(1);
    }

    // Uncomment the line below to check your results
    check_task_2(4, host_a);

    /**
     * ~TODO 5~
     * Execute kernel_thread_id kernel and then copy from 
     * the device to the host;
     */

    // Uncomment the line below to check your results

    // TODO 6: Free the memory
    kernel_thread_id<<<NMAX / 4, 4>>>(device_a, NMAX);

	err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        perror("hipDeviceSynchronize");
        exit(1);
    }

	err = hipMemcpy(host_a, device_a, NMAX * sizeof(*host_a),
		hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        perror("hipMemcpy device_a -> host_a");
        exit(1);
    }

    check_task_2(5, host_a);
	// TODO 6: Free the memory
	free(host_a);
	err = hipFree(device_a);
    if (err != hipSuccess) {
        perror("hipFree device_a");
        exit(1);
    }
    
    return 0;
}