
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define BUF_2M		(2 * 1024 * 1024)
#define BUF_32M		(32 * 1024 * 1024)

int main(void) {
    hipSetDevice(0);

    int *host_array_a = 0;
    int *host_array_b = 0;

    int *device_array_a = 0;
    int *device_array_b = 0;
    int *device_array_c = 0;

    hipError_t err;
	int i;

    // TODO 1: Allocate the host's arrays:
    // host_array_a => 32M
    // host_array_b => 32M
    host_array_a = (int*)malloc(BUF_32M * sizeof(*host_array_a));
	host_array_b = (int*)malloc(BUF_32M * sizeof(*host_array_b));

    // TODO 2: Allocate the host's arrays:
    // device_array_a => 32M
    // device_array_b => 32M
    // device_array_c => 2M
	hipMalloc(&device_array_a, BUF_32M * sizeof(*device_array_a));
	hipMalloc(&device_array_b, BUF_32M * sizeof(*device_array_b));
	hipMalloc(&device_array_c, BUF_2M * sizeof(*device_array_c));

    // Check for allocation errors
    if (host_array_a == 0 || host_array_b == 0 || 
        device_array_a == 0 || device_array_b == 0 || 
        device_array_c == 0) {
        printf("[*] Error!\n");
        return 1;
    }

    for (int i = 0; i < BUF_32M; ++i) {
        host_array_a[i] = i % 32;
        host_array_b[i] = i % 2;
    }

    printf("Before swap:\n");
    printf("a[i]\tb[i]\n");
    for (int i = 0; i < 10; ++i) {
        printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
    }

    // TODO 3: Copy from host to device
	err = hipMemcpy(device_array_a, host_array_a, BUF_32M * sizeof(*device_array_a), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        perror("cudaMemcpy(host_array_a)");
        return 1;
    }

	err = hipMemcpy(device_array_b, host_array_b, BUF_32M * sizeof(*device_array_b), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        perror("cudaMemcpy(host_array_b)");
        return 1;
    }

    // TODO 4: Swap the buffers (BUF_2M values each iteration)
    // Hint 1: device_array_c should be used as a temporary buffer
    // Hint 2: cudaMemcpy
    for (i = 0; i != BUF_32M; i += BUF_2M)
	{
		err = hipMemcpy(device_array_c, device_array_b + i, BUF_2M * sizeof(*device_array_b), hipMemcpyDeviceToDevice);
        if (err != hipSuccess) {
            perror("cudaMemcpy(device_array_b)");
            return 1;
        }

		err = hipMemcpy(device_array_b + i, device_array_a + i, BUF_2M * sizeof(*device_array_a), hipMemcpyDeviceToDevice);
        if (err != hipSuccess) {
            perror("cudaMemcpy(device_array_a)");
            return 1;
        }

		err = hipMemcpy(device_array_a + i, device_array_c,BUF_2M * sizeof(*device_array_c), hipMemcpyDeviceToDevice);
        if (err != hipSuccess) {
            perror("cudaMemcpy(device_array_c)");
            return 1;
        }
	}

    // TODO 5: Copy from device to host
    err = hipMemcpy(host_array_a, device_array_a, BUF_32M * sizeof(*host_array_a), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        perror("cudaMemcpy(host_array_a)");
        return 1;
    }

	err = hipMemcpy(host_array_b, device_array_b, BUF_32M * sizeof(*host_array_b), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        perror("cudaMemcpy(host_array_b)");
        return 1;
    }

	printf("\nAfter swap:\n");
	printf("a[i]\tb[i]\n");
	for (int i = 0; i < 10; ++i) {
		printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
	}


    printf("\nAfter swap:\n");
    printf("a[i]\tb[i]\n");
    for (int i = 0; i < 10; ++i) {
        printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
    }

    // TODO 6: Free the memory
    free(host_array_a);
	free(host_array_b);

	err = hipFree(device_array_a);
    if (err != hipSuccess) {
        perror("cudaFree(device_array_a)");
        return 1;
    }

	err = hipFree(device_array_b);
    if (err != hipSuccess) {
        perror("cudaFree(device_array_b)");
        return 1;
    }

	err = hipFree(device_array_c);
    if (err != hipSuccess) {
        perror("cudaFree(device_array_c)");
        return 1;
    }

    return 0;
}