#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "utils.h"

// TODO 6: Write the code to add the two arrays element by element and 
// store the result in another array
__global__ void add_arrays(const float *a, const float *b, float *c, int N) {
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N)
	{
		c[i] = a[i] + b[i];
	}
}

int main(void) {
    hipSetDevice(0);
    int N = 1 << 20;

    float *host_array_a = NULL;
    float *host_array_b = NULL;
    float *host_array_c = NULL;

    float *device_array_a = NULL;
    float *device_array_b = NULL;
    float *device_array_c = NULL;

	hipError_t err;

	const size_t block_size = 256;
	size_t num_blocks;

    // TODO 1: Allocate the host's arrays
    host_array_a = (float*)malloc(N * sizeof(*host_array_a));
	if (host_array_a == NULL){
		perror("malloc(host_array_a) failed\n");
		return 1;
	}

	host_array_b = (float*)malloc(N * sizeof(*host_array_b));
	if (host_array_b == NULL){
		perror("malloc(host_array_b) failed\n");
		return 1;
	}

	host_array_c = (float*)malloc(N * sizeof(*host_array_c));
	if (host_array_c == NULL){
		perror("malloc(host_array_c) failed\n");
		return 1;
	}
	
	// TODO 2: Allocate the device's arrays
    int err = hipMalloc(&device_array_a, N * sizeof(*device_array_a));
	if (err != hipSuccess || device_array_a == NULL)
	{
		perror("hipMalloc(device_array_a) failed\n");
		return 1;
	}

	err = hipMalloc(&device_array_b, N * sizeof(*device_array_b));
	if (err != hipSuccess || device_array_b == NULL)
	{
		perror("hipMalloc(device_array_b) failed\n");
		return 1;
	}

	err = hipMalloc(&device_array_c, N * sizeof(*device_array_c));
	if (err != hipSuccess || device_array_c == NULL)
	{
		perror("hipMalloc(device_array_c) failed\n");
		return 1;
	}

    // TODO 3: Check for allocation errors


    // TODO 4: Fill array with values; use fill_array_float to fill
    // host_array_a and fill_array_random to fill host_array_b. Each
    // function has the signature (float *a, int n), where n = the size.
	fill_array_float(host_array_a, N);
	fill_array_random(host_array_b, N);
    
    // TODO 5: Copy the host's arrays to device
    err = hipMemcpy(device_array_a, host_array_a, N * sizeof(*host_array_a), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		perror("hipMemcpy(host_array_a) failed\n");
		return 1;
	}

	err = hipMemcpy(device_array_b, host_array_b, N * sizeof(*host_array_b), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		perror("hipMemcpy(host_array_b) failed\n");
		return 1;
	}


    // TODO 6: Execute the kernel, calculating first the grid size
    // and the amount of threads in each block from the grid
	num_blocks = N / block_size;

	if (N % block_size)
	{
		++num_blocks;
	}

	add_arrays<<<num_blocks, block_size>>>(device_array_a, device_array_b, device_array_c, N);

	err = hipDeviceSynchronize();
	if (err != hipSuccess)
	{
		perror("hipDeviceSynchronize failed\n");
		return 1;
	}

    // TODO 7: Copy back the results and then uncomment the checking function
	err = hipMemcpy(host_array_c, device_array_c,
		N * sizeof(*host_array_c), hipMemcpyDeviceToHost);
	if (err != hipSuccess)
		{
			perror("hipMemcpy(host_array_c) failed\n");
			return 1;
		}

    check_task_3(host_array_a, host_array_b, host_array_c, N);

    // TODO 8: Free the memory
	free(host_array_a);
	free(host_array_b);
	free(host_array_c);

	err = hipFree(device_array_a);
	if (err != hipSuccess)
	{
		perror("hipFree(device_array_a) failed\n");
		return 1;
	}

	err = hipFree(device_array_b);
	if (err != hipSuccess)
	{
		perror("hipFree(device_array_b) failed\n");
		return 1;
	}

	err = hipFree(device_array_c);
	if (err != hipSuccess)
	{
		perror("hipFree(device_array_c) failed\n");
		return 1;
	}

    return 0;
}