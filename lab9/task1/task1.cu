#include "hip/hip_runtime.h"
#include <stdio.h>

#define NUM_ELEM        8
#define NUM_THREADS     10

__global__ void childKernel() {
    printf("Hello ");
}

__global__ void parentKernel() {
    // launch child
    childKernel<<<1,1>>>();
    if (hipSuccess != hipGetLastError()) {
        return;
    }
    
    // wait for child to complete
    if (hipSuccess != hipDeviceSynchronize()) {
        return;
    }
    
    printf("World!\n");
}

__global__ void concurrentRW(int *data) {
    // NUM_THREADS try to read and write at same location
    //data[blockIdx.x] = data[blockIdx.x] + threadIdx.x;
    atomicAdd(&data[blockIdx.x], threadIdx.x);
}

int main(int argc, char *argv[]) {
    // launch parent
    parentKernel<<<1,1>>>();
    if (hipSuccess != hipGetLastError()) {
        return 1;
    }
    
    // wait for parent to complete
    if (hipSuccess != hipDeviceSynchronize()) {
        return 2;
    }

    int* data = NULL;
    bool errorsDetected = false;

    hipMallocManaged(&data, NUM_ELEM * sizeof(unsigned long long int));
    if (data == 0) {
        cout << "[HOST] Couldn't allocate memory\n";
        return 1;
    }

    // init all elements to 0
    hipMemset(data, 0, NUM_ELEM);

    // launch kernel writes
    concurrentRW<<<NUM_ELEM, NUM_THREADS>>>(data);
    hipDeviceSynchronize();
    if (hipSuccess != hipGetLastError()) {
        return 1;
    }

    for(int i = 0; i < NUM_ELEM; i++) {
        cout << i << ". " << data[i] << endl;
        if(data[i] != (NUM_THREADS * (NUM_THREADS - 1) / 2)) {
            errorsDetected = true;
        }
    }

    if(errorsDetected) {
        cout << "Errors detected" << endl;
    } else {
        cout << "OK" << endl;
    }
    
    return 0;
}